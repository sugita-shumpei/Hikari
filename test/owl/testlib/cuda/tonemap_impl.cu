#include "hip/hip_runtime.h"
#include "tonemap_impl.h"
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#define EPS 0.0001f

__global__ void HikariTestOwlTonemap_estimateLuminanceImpl(int width, int height, const float3* input_buffer, float* luminance_buffer, float* luminance_log_buffer)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= width || y >= height) { return; }
	auto input_color = input_buffer[width * y + x];
	auto luminance   = 0.2126f * input_color.x + 0.7152f * input_color.y + 0.0722f * input_color.z;
	luminance_buffer[width * y + x]     = luminance;
	luminance_log_buffer[width * y + x] = logf(EPS + luminance);
}

__global__ void HikariTestOwlTonemap_tonemapColorRGBA8Impl(
	int           width , 
	int           height, 
	const float3* input_buffer, 
	unsigned int* output_buffer, 
	float         max_luminance, 
	float         ave_luminance,
	float         key_value)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= width || y >= height) { return; }
	auto input_color       = input_buffer[width * y + x];
	// auto luminance         = 0.2126f * input_color.x + 0.7152f * input_color.y + 0.0722f * input_color.z;
	// auto crr_luminance     = key_value *     luminance / ave_luminance;
	// auto crr_max_luminance = key_value * max_luminance / ave_luminance;
	input_color.x *= (key_value / ave_luminance);
	input_color.y *= (key_value / ave_luminance);
	input_color.z *= (key_value / ave_luminance);
	input_color.x = input_color.x / (1.0f + input_color.x);
	input_color.y = input_color.y / (1.0f + input_color.y);
	input_color.z = input_color.z / (1.0f + input_color.z);
	unsigned int r = fminf(255, fmaxf(0, int(input_color.x * 256.f)));
	unsigned int g = fminf(255, fmaxf(0, int(input_color.y * 256.f)));
	unsigned int b = fminf(255, fmaxf(0, int(input_color.z * 256.f)));
	output_buffer[width * y + x] = (r << 0u) + (g << 8u) + (b << 16u) + (0xffu << 24u);
}

void HikariTestOwlTonemap_estimateLuminance(hipStream_t  stream, int width, int height, const float3* input_buffer, float* luminance_buffer, float* luminance_log_buffer)
{
	const int block_size_x = 32;
	const int block_size_y = 32;
	const int grid_size_x = (width + block_size_x - 1) / block_size_x;
	const int grid_size_y = (height + block_size_y - 1) / block_size_y;

	dim3 grid(grid_size_x, grid_size_y, 1);
	dim3 threads(block_size_x, block_size_y, 1);
	HikariTestOwlTonemap_estimateLuminanceImpl << <grid, threads , 0, stream>> > (width, height, input_buffer, luminance_buffer, luminance_log_buffer);
}

void HikariTestOwlTonemap_estimateMaxAndAverage(hipStream_t  stream, int width, int height, const float* luminance_buffer, const float* luminance_log_buffer, float* p_max_luminance, float* p_ave_luminance)
{
	auto luminance_buffer_ptr     = thrust::device_pointer_cast(luminance_buffer);
	auto luminance_log_buffer_ptr = thrust::device_pointer_cast(luminance_log_buffer);
	auto  max_iter  = thrust::max_element(thrust::cuda::par.on(stream),luminance_buffer_ptr, luminance_buffer_ptr + width * height);
	float max_value = 0.0f;
	hipMemcpyAsync(&max_value, max_iter.get(), sizeof(float), hipMemcpyDeviceToHost,stream);
	auto ave_value = expf(thrust::reduce(luminance_log_buffer_ptr, luminance_log_buffer_ptr + width * height) / (width * height));
	printf("value=%f %f\n", ave_value, max_value);
	if (p_ave_luminance) *p_ave_luminance = ave_value;
	if (p_max_luminance) *p_max_luminance = max_value;
}

void HikariTestOwlTonemap_tonemapColorRGBA8(hipStream_t  stream, int width, int height, const float3* input_buffer, unsigned int* output_buffer, float max_luminance, float ave_luminance, float key_value)
{
	const int block_size_x = 32;
	const int block_size_y = 32;
	const int grid_size_x  = (width  + block_size_x - 1) / block_size_x;
	const int grid_size_y  = (height + block_size_y - 1) / block_size_y;
	
	dim3 grid(grid_size_x, grid_size_y,1);
	dim3 threads(block_size_x, block_size_y, 1);
	HikariTestOwlTonemap_tonemapColorRGBA8Impl<<<grid, threads, 0,stream >>>(width,height,input_buffer,output_buffer, max_luminance,ave_luminance, key_value);
}
