#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <owl/owl_device.h>
#include <owl/common/owl-common.h>
#include <owl/common/math/random.h>
#include "deviceCode.h"
 
extern "C" { __constant__ LaunchParams optixLaunchParams; }

struct PayloadData {
	owl::vec3f   color;
};

__forceinline__ __device__ owl::vec2f normalize_uv(owl::vec2f vt) {
	vt.x = fmodf(vt.x, 1.0f);
	vt.y = fmodf(vt.y, 1.0f);
	if (vt.x < 0.0f) {
		vt.x = vt.x + 1.0f;
	}
	if (vt.y < 0.0f) {
		vt.y = vt.y + 1.0f;
	}
	return vt;
}

OPTIX_RAYGEN_PROGRAM(simpleRG)() {
	const owl::vec2i idx = owl::getLaunchIndex();
	const owl::vec2i dim = owl::getLaunchDims();
	auto& rg_data        = owl::getProgramData<RayGenData>();

	owl::LCG<24>           random;
	random.init(dim.x * idx.y + idx.x, optixLaunchParams.accum_sample);

	constexpr auto frame_samples = 1;

	auto color = owl::vec3f(0.0f,0.0f,0.0f);
	for (int i = 0; i < frame_samples; ++i) {
		float px      = ((float)idx.x + random() - 0.5f) / ((float)dim.x);
		float py      = ((float)idx.y + random() - 0.5f) / ((float)dim.y);
		auto  ray_dir = rg_data.camera.getRayDirection(px, py);

		owl::RayT<0, 1> ray(rg_data.camera.eye,
			owl::normalize(ray_dir),
			rg_data.min_depth, rg_data.max_depth
		);

		auto  payload = PayloadData();
		payload.color = owl::vec3f(0.0f, 0.0f, 0.0f);
		owl::trace(rg_data.world, ray, 1, payload);
		color += payload.color;
	}


	auto res = optixLaunchParams.accum_buffer[dim.x * idx.y + idx.x];
	auto col = (color + owl::vec3f(res));
	auto smp = res.w + frame_samples;
	optixLaunchParams.accum_buffer[dim.x * idx.y + idx.x] = owl::vec4f(col, smp);
	col *= (1.0f / smp);
	rg_data.fb_data[dim.x * idx.y + idx.x] = owl::make_rgba(col);
}

OPTIX_MISS_PROGRAM(  simpleMS)() {
	auto& payload = owl::getPRD<PayloadData>();
	auto& ms_data = owl::getProgramData<MissProgData>();
	auto  ray_dir = owl::normalize(owl::vec3f(optixGetWorldRayDirection()));
	float phi     = atan2f(ray_dir.z, ray_dir.x);
	float theta   = asinf (ray_dir.y)  ;
	float x       = (phi   / M_PI + 1.0f)*0.5f;
	float y       = (theta / M_PI + 0.5f);

	// x y z
	//x=(ray_dir.x + 1.0f)*0.5f
	//y=(ray_dir.y + 1.0f)*0.5f
	auto tmp_col = tex2D<float4>(ms_data.texture_envlight ,x, y);
	payload.color.x = tmp_col.x;
	payload.color.y = tmp_col.y;
	payload.color.z = tmp_col.z;
}

OPTIX_ANY_HIT_PROGRAM(simpleAH)() {
	auto& ch_data = owl::getProgramData<HitgroupData>();
	if (optixIsTriangleHit()) {
		auto pri_idx = optixGetPrimitiveIndex();
		auto tri_idx = ch_data.indices[pri_idx];
		auto bary    = optixGetTriangleBarycentrics();

		auto vt0 = ch_data.uvs[tri_idx.x];
		auto vt1 = ch_data.uvs[tri_idx.y];
		auto vt2 = ch_data.uvs[tri_idx.z];

		auto vt = normalize_uv((1.0f - (bary.x + bary.y)) * vt0 + bary.x * vt1 + bary.y * vt2);
		auto tmp_col = tex2D<float4>(ch_data.texture_alpha, vt.x, vt.y);
		if (tmp_col.w * tmp_col.x * tmp_col.y* tmp_col.z < 0.5f) {
			optixIgnoreIntersection();
		}
	}
}

OPTIX_CLOSEST_HIT_PROGRAM(simpleCH)() {
	auto& ch_data = owl::getProgramData<HitgroupData>();
	auto& payload = owl::getPRD<PayloadData>();

	auto pri_idx = optixGetPrimitiveIndex();
	auto tri_idx = ch_data.indices[pri_idx];
	auto v0 = ch_data.vertices[tri_idx.x];
	auto v1 = ch_data.vertices[tri_idx.y];
	auto v2 = ch_data.vertices[tri_idx.z];
	auto v01 = v1 - v0;
	auto v12 = v2 - v1;
	auto f_normal = owl::normalize(owl::cross(v01, v12));

	auto bary = optixGetTriangleBarycentrics();
	auto vn0 = ch_data.normals[tri_idx.x];
	auto vn1 = ch_data.normals[tri_idx.y];
	auto vn2 = ch_data.normals[tri_idx.z];

	auto vt0 = ch_data.uvs[tri_idx.x];
	auto vt1 = ch_data.uvs[tri_idx.y];
	auto vt2 = ch_data.uvs[tri_idx.z];

	auto vt = normalize_uv((1.0f - (bary.x + bary.y)) * vt0 + bary.x * vt1 + bary.y * vt2);

	auto vn0_l = owl::dot(vn0, vn0);
	auto vn1_l = owl::dot(vn1, vn1);
	auto vn2_l = owl::dot(vn2, vn2);
	if (vn0_l < 0.01f) { vn0 = f_normal; }
	if (vn1_l < 0.01f) { vn1 = f_normal; }
	if (vn2_l < 0.01f) { vn2 = f_normal; }
	// 
	auto v_normal = owl::normalize((1.0f - (bary.x + bary.y)) * vn0 + bary.x * vn1 + bary.y * vn2);
	//payload.color.x = (v_normal.x + 1.0f) * 0.5f;
	//payload.color.y = (v_normal.y + 1.0f) * 0.5f;
	//payload.color.z = (v_normal.z + 1.0f) * 0.5f;
	auto vtg0 = owl::vec3f(ch_data.tangents[tri_idx.x]);
	auto vtg1 = owl::vec3f(ch_data.tangents[tri_idx.y]);
	auto vtg2 = owl::vec3f(ch_data.tangents[tri_idx.z]);

	auto vbs0 = ch_data.tangents[tri_idx.x].w;
	auto vbs1 = ch_data.tangents[tri_idx.y].w;
	auto vbs2 = ch_data.tangents[tri_idx.z].w;
	auto vbn0 =  vbs0 * owl::normalize(owl::cross(vn0, vtg0));
	auto vbn1 =  vbs1 * owl::normalize(owl::cross(vn1, vtg1));
	auto vbn2 =  vbs2 * owl::normalize(owl::cross(vn2, vtg2));
	auto v_binormal   = owl::normalize((1.0f - (bary.x + bary.y)) * vbn0 + bary.x * vbn1 + bary.y * vbn2);
	//payload.color.x = (v_binormal.x + 1.0f) * 0.5f;
	//payload.color.y = (v_binormal.y + 1.0f) * 0.5f;
	//payload.color.z = (v_binormal.z + 1.0f) * 0.5f;

	auto v_tangent = owl::normalize(owl::cross(v_binormal, v_normal));

	auto tmp_bump = tex2D<float4>(ch_data.texture_normal, vt.x, vt.y);
	// shading�@��(�����܂ŕ`��p)
	auto fin_normal = owl::normalize(tmp_bump.z * v_normal + (2.0f * tmp_bump.x - 1.0f) * v_tangent + (2.0f * tmp_bump.y - 1.0f) * v_binormal);

	//payload.color.x = (fin_normal.x + 1.0f) * 0.5f;
	//payload.color.y = (fin_normal.y + 1.0f) * 0.5f;
	//payload.color.z = (fin_normal.z + 1.0f) * 0.5f;

	auto ambient_col = ch_data.color_ambient *owl::vec3f(tex2D<float4>(ch_data.texture_ambient, vt.x, vt.y));

	//color = ch_data.colors[pri_idx];
	payload.color.x = ambient_col.x;
	payload.color.y = ambient_col.y;
	payload.color.z = ambient_col.z;
}

OPTIX_DIRECT_CALLABLE_PROGRAM(simpleDC1)(owl::vec4f& c) {
	auto& callable_data  = owl::getProgramData<CallableData>();
	const owl::vec2i idx = owl::getLaunchIndex();
	const owl::vec2i dim = owl::getLaunchDims();
	c = callable_data.color;
}

OPTIX_DIRECT_CALLABLE_PROGRAM(simpleDC2)(owl::vec4f& c) {
	auto& callable_data  = owl::getProgramData<CallableData>();
	const owl::vec2i idx = owl::getLaunchIndex();
	const owl::vec2i dim = owl::getLaunchDims();
	c = callable_data.color;
}
