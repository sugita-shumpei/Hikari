#include "hip/hip_runtime.h"

#include <optix_device.h>
#include <owl/owl_device.h>
#include <owl/common/owl-common.h>
#include <owl/common/math/random.h>
#include <owl/common/math/vec.h>
#include "deviceCode.h"

extern "C" { __constant__ LaunchParams optixLaunchParams;  }

struct PayloadData {
  owl::vec3f   s_normal;
  unsigned int surface_idx;
  owl::vec3f   g_normal;
  float        distance;
  owl::vec2f   texcoord;
};
__forceinline__ __device__ void       traceRadiance(const owl::RayT<0, 1>& ray, PayloadData& payload) {
  owl::trace(optixLaunchParams.tlas, ray, RAY_TYPE_COUNT, payload, RAY_TYPE_RADIANCE);
}
__forceinline__ __device__ bool       traceOccluded(const owl::RayT<0, 1>& ray) {
  unsigned int occluded = 0;
  optixTrace(optixLaunchParams.tlas, { ray.origin.x,ray.origin.y,ray.origin.z }, { ray.direction.x,ray.direction.y,ray.direction.z }, 0.0f, 1e10f, 0.0f, 255u,
    OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
    RAY_TYPE_OCCLUDED,
    RAY_TYPE_COUNT,
    RAY_TYPE_OCCLUDED, occluded);
  return occluded;
}
__forceinline__ __device__ float      fresnel1(float eta, float cos_in_2, float cos_out_2) {
  float cos_in = sqrtf(cos_in_2);
  float cos_out= sqrtf(cos_out_2);
  float rp = (eta * cos_in - cos_out) / (eta * cos_in + cos_out);
  float rs = (eta * cos_out - cos_in) / (eta * cos_out + cos_in);
  return 0.5f * (rp * rp + rs * rs);
}
__forceinline__ __device__ float      fresnel2(float eta, float k       , float cos_in_2, float cos_out_2) {

  float cos_in  = sqrtf(cos_in_2);
  float cos_out = sqrtf(cos_out_2);
  float rp1     = (eta * cos_in - cos_out);
  float rp2     = (eta * cos_in + cos_out);
  float rpk     = (k * cos_in);
  float rs1     = (eta * cos_out - cos_in);
  float rs2     = (eta * cos_out + cos_in);
  float rsk     = (k * cos_out);
  float rp      = (rp1*rp1 + rpk*rpk)/ (rp2 * rp2 + rpk * rpk);
  float rs      = (rs1*rs1 + rsk*rsk)/ (rs2 * rs2 + rsk * rsk);
  return 0.5f * (rp + rs);
}
__forceinline__ __device__ owl::vec3f fresnel2(const owl::vec3f& eta, const owl::vec3f& k, float cos_in_2, const owl::vec3f& cos_out_2) {
  return owl::vec3f(fresnel2(eta.x, k.x, cos_in_2, cos_out_2.x), fresnel2(eta.y, k.y, cos_in_2, cos_out_2.y), fresnel2(eta.z, k.z, cos_in_2, cos_out_2.z));
}

__forceinline__ __device__ owl::vec3f random_in_pdf_cosine(owl::LCG<24>& random) {
  float cos_tht = sqrtf(1 - random());
  float sin_tht = sqrtf(fmaxf(1 - cos_tht * cos_tht,0.0f));
  float phi = 2.0f * M_PI * random();
  float cos_phi = cosf(phi);
  float sin_phi = sinf(phi);
  return { sin_tht * cos_phi,sin_tht * sin_phi,cos_tht };
}

__forceinline__ __device__ bool       traceOccluded(const owl::RayT<RAY_TYPE_OCCLUDED, RAY_TYPE_COUNT>& ray) {
  unsigned int occluded = 0;
  optixTrace(optixLaunchParams.tlas, { ray.origin.x,ray.origin.y,ray.origin.z }, { ray.direction.x,ray.direction.y,ray.direction.z }, 0.0f, 1e10f, 0.0f, 255u,
    OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
    RAY_TYPE_OCCLUDED,
    RAY_TYPE_COUNT,
    RAY_TYPE_OCCLUDED, occluded);
  return occluded;
}

__forceinline__ __device__ bool       shade_material(
  const PayloadData& payload,
  float              min_depth,
  float              max_depth,
  owl::vec3f&        ray_org,
  owl::vec3f&        ray_dir,
  owl::vec3f&        color,
  owl::vec3f&        throughput,
  owl::LCG<24>&      random) {
  ray_org           = ray_org + payload.distance * ray_dir;
  auto s_normal     = payload.s_normal;
  auto g_normal     = payload.g_normal;
  auto s_cosine_in  = -owl::dot(s_normal, ray_dir);
  auto g_cosine_in  = -owl::dot(g_normal, ray_dir);

  // CLOSEST HIT
  if (payload.surface_idx > 0) {
    auto surface_idx = payload.surface_idx - 1;
    // Surfaceを取得
    auto& surface    = optixLaunchParams.surfaces[surface_idx];
    // Light
    if ((surface.type & SURFACE_TYPE_MASK) == SURFACE_TYPE_DIFFUSE   ) { // DIFFUSEはおおむね一致。
      if (s_cosine_in < 0.0f || g_cosine_in < 0.0f) { return true; }
      Onb  onb(s_normal);
      auto refl_dir      = onb.local(random_in_pdf_cosine(random));
      float g_cosine_out = owl::dot(g_normal, refl_dir);
      if   (g_cosine_out < 0.0f) { return true; }

      auto diffuse = surface.loadDiffuse(optixLaunchParams.textures, payload.texcoord.x, payload.texcoord.y);
      ray_org           += 0.01f * s_normal;
      ray_dir            = refl_dir;
      throughput        *= diffuse.reflectance;
      return false;
    }
    if ((surface.type & SURFACE_TYPE_MASK) == SURFACE_TYPE_CONDUCTOR ) { // PLASTICはおおむね一致。
      if (s_cosine_in < 0.0f || g_cosine_in < 0.0f) { return true; }
      auto refl_dir     = owl::normalize(ray_dir + 2.0f * s_cosine_in * s_normal);
      float g_cosine_out= owl::dot(g_normal, refl_dir);
      if (g_cosine_out < 0.0f) { return true; }

      auto conductor    = surface.loadConductor(optixLaunchParams.textures, payload.texcoord.x, payload.texcoord.y);

      auto cos_1_in     = s_cosine_in;
      auto cos_1_in_sq  = cos_1_in * cos_1_in;
      auto sin_1_in_sq  = 1.0f - cos_1_in_sq;
      auto sin_1_out_sq = owl::vec3f(sin_1_in_sq) / (conductor.eta * conductor.eta);
      auto cos_1_out_sq = owl::vec3f(fmaxf(1.0f - sin_1_out_sq.x, 0.0f), fmaxf(1.0f - sin_1_out_sq.y, 0.0f), fmaxf(1.0f - sin_1_out_sq.z, 0.0f));
      auto r0           = fresnel2(conductor.eta, conductor.k, cos_1_in_sq, cos_1_out_sq);
      
      ray_org          += 0.01f * s_normal;
      ray_dir           = refl_dir;
      throughput       *= r0 * conductor.specular_reflectance;
      return false;
    }
    if ((surface.type & SURFACE_TYPE_MASK) == SURFACE_TYPE_DIELECTRIC) { // PLASTICはおおむね一致。
      auto  dielectric  = surface.loadDielectric(optixLaunchParams.textures, payload.texcoord.x, payload.texcoord.y);
      float eta         = s_cosine_in > 0.0f ? dielectric.eta : 1.0f / dielectric.eta;
      auto  r_normal    = s_cosine_in > 0.0f ? s_normal : -s_normal;

      auto cos_1_in     = s_cosine_in;
      auto cos_1_in_sq  = cos_1_in * cos_1_in;
      auto sin_1_in_sq  = 1.0f - cos_1_in_sq;
      auto sin_1_out_sq = sin_1_in_sq / (eta* eta);
      auto cos_1_out_sq = fmaxf(1.0f - sin_1_out_sq, 0.0f);
      auto r0           = fresnel1(eta, cos_1_in_sq, cos_1_out_sq);

      if (random() < r0) {
        auto refl_dir   = owl::normalize(ray_dir + 2.0f * s_cosine_in * s_normal);
        ray_org        += 0.01f * r_normal;
        ray_dir         = refl_dir;
        throughput     *= dielectric.specular_reflectance;
      }
      else {
        auto tran_dir  = owl::normalize((ray_dir + s_cosine_in * s_normal) / eta - sqrtf(1.0f - sin_1_out_sq) * r_normal);
        ray_org       -= 0.01f * r_normal;
        ray_dir        = tran_dir;
        throughput    *= dielectric.specular_transmittance/(eta*eta);
      }
      return false;
    }
    if ((surface.type & SURFACE_TYPE_MASK) == SURFACE_TYPE_PLASTIC   ) { // PLASTICはおおむね一致。
      if (s_cosine_in < 0.0f || g_cosine_in < 0.0f) { return true; }
      auto plastic                         = surface.loadPlastic(optixLaunchParams.textures, payload.texcoord.x, payload.texcoord.y);
      bool nonlinear                       = plastic.int_fresnel_diffuse_reflectance > 0.0f;
      auto int_fresnel_diffuse_reflectance = fabsf(plastic.int_fresnel_diffuse_reflectance);
      auto diff_reflectance_fact           = nonlinear ? plastic.diffuse_reflectance : owl::vec3f(1.0f);
      auto diff_crr                        = owl::vec3f(1.0f) - (diff_reflectance_fact * int_fresnel_diffuse_reflectance);

      auto cos_1_in     = s_cosine_in;
      auto cos_1_in_sq  = cos_1_in * cos_1_in;
      auto sin_1_in_sq  = 1.0f - cos_1_in_sq;
      auto sin_1_out_sq = sin_1_in_sq / (plastic.eta * plastic.eta);
      auto cos_1_out_sq = fmaxf(1.0f - sin_1_out_sq, 0.0f);

      auto r0 = fresnel1(plastic.eta, cos_1_in_sq, cos_1_out_sq);
      auto t0 = 1.0f - r0;

      auto spec_refl_dir = owl::normalize(ray_dir + 2.0f * s_cosine_in * s_normal);
      auto diff_refl_dir = random_in_pdf_cosine(random);

      auto cos_2_in     = diff_refl_dir.z;
      auto cos_2_in_sq  = cos_2_in * cos_2_in;
      auto sin_2_in_sq  = 1.0f - cos_2_in_sq;
      auto sin_2_out_sq = sin_2_in_sq / (plastic.eta * plastic.eta);
      auto cos_2_out_sq = fmaxf(1.0f - sin_2_out_sq, 0.0f);

      Onb onb(s_normal);
      diff_refl_dir = onb.local(diff_refl_dir);

      auto r1 = fresnel1(plastic.eta, cos_2_in_sq, cos_2_out_sq);
      auto t1 = 1.0f - r1;

      auto spec_g_cos_out = owl::dot(spec_refl_dir, g_normal);
      auto diff_g_cos_out = owl::dot(diff_refl_dir, g_normal);

      auto total_spec_reflectance =  r0 * static_cast<float>(spec_g_cos_out > 0.0f) * plastic.specular_reflectance;
      auto total_diff_reflectance = (t0 * t1 / plastic.eta * plastic.eta) * static_cast<float>(diff_g_cos_out > 0.0f) * (plastic.diffuse_reflectance / diff_crr);

      auto ave_total_spec_reflectance = owl::dot(total_spec_reflectance, owl::vec3f(1.0f)) / 3.0f;
      auto ave_total_diff_reflectance = owl::dot(total_diff_reflectance, owl::vec3f(1.0f)) / 3.0f;

      auto sum_ave_reflectance = (ave_total_spec_reflectance + ave_total_diff_reflectance);
      if ( sum_ave_reflectance <= 0.0f) { return true; }

      auto prob = (ave_total_spec_reflectance) / sum_ave_reflectance;

      ray_org      += 0.01f * s_normal;
      if (random() < prob)
      {
        ray_dir     = spec_refl_dir;
        throughput *= total_spec_reflectance/prob;
      }
      else
      {
        ray_dir     = diff_refl_dir;
        throughput *= total_diff_reflectance/(1.0f-prob);
      }
      return false;
    }
    return false;
  }
  // MISS
  else
  {
    color += throughput * optixLaunchParams.light.envmap.sample(ray_dir);
    return true;
  }
}
// 実際の描画処理はここで実行
OPTIX_RAYGEN_PROGRAM(default)() {
  const owl::vec2i idx = owl::getLaunchIndex();
  const owl::vec2i dim = owl::getLaunchDims();
  const SBTRaygenData& sbt_rg_data = owl::getProgramData<SBTRaygenData>();

  auto frame_index = dim.x * idx.y + idx.x;

  constexpr auto frame_samples = 1;
  constexpr auto trace_depth   = 3;

  auto payload = PayloadData();
  owl::LCG<24> random = {};
  random.init(frame_index, sbt_rg_data.sample);

  auto color = owl::vec3f(0.0f, 0.0f, 0.0f);
  for (int i = 0; i < frame_samples; ++i) {
    payload = PayloadData();
    auto uv = owl::vec2f(
      2.0f * (((float)idx.x + random()) / (float)sbt_rg_data.width )-1.0f,
      2.0f * (((float)idx.y + random()) / (float)sbt_rg_data.height)-1.0f
    );

    auto ray_org    = sbt_rg_data.camera.eye;
    auto ray_dir    = (sbt_rg_data.camera.dir_w + uv.x * sbt_rg_data.camera.dir_u + uv.y * sbt_rg_data.camera.dir_v);

    auto throughput = owl::vec3f(1.0f, 1.0f, 1.0f);
    bool done       = false;
    for (int j = 0; (j < trace_depth) && !done; ++j) {
      float tmin = (j == 0) ? sbt_rg_data.camera.near_clip : 0.01f;
      float tmax = (j == 0) ? sbt_rg_data.camera.far_clip  : 1e11f;
      owl::RayT<0, 1> ray(ray_org, ray_dir, tmin, tmax);
      traceRadiance(ray, payload);
      done   = shade_material(payload, 0.01f, 1e11f, ray_org, ray_dir, color, throughput, random);

    }
  }

  auto prv_accum = owl::vec3f(sbt_rg_data.accum_buffer[frame_index]);
  auto cur_accum = prv_accum + color;
  auto cur_frame = cur_accum / static_cast<float>(sbt_rg_data.sample + frame_samples);
  sbt_rg_data.accum_buffer[frame_index] = make_float3(cur_accum.x, cur_accum.y, cur_accum.z);
  sbt_rg_data.frame_buffer[frame_index] = make_float3(cur_frame.x, cur_frame.y, cur_frame.z);
}
// レイタイプ: Radiance
// 最近傍シェーダ(サーフェス情報を取得)
OPTIX_CLOSEST_HIT_PROGRAM(default_triangle)() {
  const SBTHitgroupData& sbt_hg_data = owl::getProgramData<SBTHitgroupData>();
  PayloadData&     payload = owl::getPRD<PayloadData>();
  auto vertex_buffer = reinterpret_cast<const float3*>(sbt_hg_data.vertex_buffer);
  auto normal_buffer = sbt_hg_data.normal_buffer;
  auto texcrd_buffer = sbt_hg_data.texcrd_buffer;
  auto index_buffer  = reinterpret_cast<const uint3*>(sbt_hg_data.index_buffer);
  auto prim_index    = optixGetPrimitiveIndex();
  auto tri_index     = index_buffer[prim_index];
  auto v0 = owl::vec3f(vertex_buffer[tri_index.x]);
  auto v1 = owl::vec3f(vertex_buffer[tri_index.y]);
  auto v2 = owl::vec3f(vertex_buffer[tri_index.z]);

  auto n0 = owl::vec3f(normal_buffer[tri_index.x]);
  auto n1 = owl::vec3f(normal_buffer[tri_index.y]);
  auto n2 = owl::vec3f(normal_buffer[tri_index.z]);

  auto t0       = owl::vec2f(texcrd_buffer[tri_index.x]);
  auto t1       = owl::vec2f(texcrd_buffer[tri_index.y]);
  auto t2       = owl::vec2f(texcrd_buffer[tri_index.z]);
  auto v01      = v1 - v0;
  auto v12      = v2 - v1;
  float2 bary   = optixGetTriangleBarycentrics();
  auto f_normal = owl::normalize(owl::cross(v01, v12));
  auto s_normal = owl::normalize((1.0f - (bary.x + bary.y)) * n0 + bary.x * n1 + bary.y * n2);

  auto vt             = (1.0f - (bary.x + bary.y)) * t0 + bary.x * t1 + bary.y * t2;
  payload.surface_idx = sbt_hg_data.surfaces + 1;
  payload.texcoord    = vt;
  payload.distance    = optixGetRayTmax();
  payload.g_normal    = f_normal;
  payload.s_normal    = s_normal;
  // 実際の処理はCallableで実行する
}
// ミスシェーダ  (サーフェス情報を取得)
OPTIX_MISS_PROGRAM(default)() {
  PayloadData& payload = owl::getPRD<PayloadData>();
  payload.surface_idx  = 0;
  payload.texcoord     = { 0.0f, 0.0f };
  payload.distance     = 0.0f;
  payload.g_normal     = { 0.0f,0.0f, 0.0f };
  payload.s_normal     = { 0.0f,0.0f, 0.0f };
}
// レイタイプ: Occluded
// 最近傍シェーダ(可視情報を取得)
OPTIX_CLOSEST_HIT_PROGRAM(occlude_triangle)() {
  optixSetPayload_0(1);
}
// ミスシェーダ  (可視情報を取得)
OPTIX_MISS_PROGRAM(occlude)() {
  optixSetPayload_0(0);
}
//// AnyHitシェーダ(αテストを起動）
//OPTIX_ANY_HIT_PROGRAM(simpleAH)() {
//  auto& ch_data = owl::getProgramData<HitgroupData>();
//  if (optixIsTriangleHit()) {
//    auto pri_idx = optixGetPrimitiveIndex();
//    auto tri_idx = ch_data.indices[pri_idx];
//    auto bary = optixGetTriangleBarycentrics();
//
//    auto vt0 = ch_data.uvs[tri_idx.x];
//    auto vt1 = ch_data.uvs[tri_idx.y];
//    auto vt2 = ch_data.uvs[tri_idx.z];
//
//    auto vt = normalize_uv((1.0f - (bary.x + bary.y)) * vt0 + bary.x * vt1 + bary.y * vt2);
//    auto tmp_col = tex2D<float4>(ch_data.texture_alpha, vt.x, vt.y);
//    if (tmp_col.w * tmp_col.x * tmp_col.y * tmp_col.z < 0.5f) {
//      optixIgnoreIntersection();
//    }
//  }
//}
